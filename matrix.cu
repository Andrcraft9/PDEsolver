#include "hip/hip_runtime.h"
#include "matrix.hpp"

#define BLOCK_SIZE 16

// Device methods

// CUDA Kernel matvec
__host__ __device__ int iDivUp(int a, int b)
{ 
    return ((a % b) != 0) ? (a / b + 1) : (a / b);
}

__forceinline__ __device__ int dev_index(int m, int n, int rows, int cols) { return m + n*rows;}
{
    return 
}

__global__ void dev_matmul(double* a, double* b, double* c, int rows, int cols)
{
    int m = blockDim.x*blockIdx.x + threadIdx.x;
    int n = blockDim.y*blockIdx.y + threadIdx.y;
    int k = dev_index(m, n, rows, cols);

    double s = 0;
    for(int i = 0; i < rows; ++i)
    {
            int ka = dev_index(m, i, rows, cols);
            int kb = dev_index(i, n, rows, cols);
            s = s + a[ka] * b[kb];
    }

    c[k] = s;
}

Matrix Matrix::device_multiply(const Matrix& a) const
{
    Matrix b(_rows, _cols, 0, this->_name + "(device)*" + a._name);

    const double *raw_ptr_this = thrust::raw_pointer_cast(this->_dev_vals.data());
    const double *raw_ptr_a = thrust::raw_pointer_cast(a._dev_vals.data());
    double *raw_ptr_b = thrust::raw_pointer_cast(b._dev_vals.data());

    dim3 DimBlock(BLOCK_SIZE, BLOCK_SIZE);
    dim3 DimGrid(iDivUp(_rows, BLOCK_SIZE), iDivUp(_cols, BLOCK_SIZE));
    dev_matmul<<<DimBlock, DimGrid>>>(raw_ptr_this, raw_ptr_a, raw_ptr_b, _rows, _cols);
    
    //thrust::plus<double> op;
    //thrust::transform(_dev_vals.begin(), _dev_vals.end(), a._dev_vals.begin(), b._dev_vals.begin(), op);
}

